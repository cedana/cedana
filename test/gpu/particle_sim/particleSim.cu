#include <stdio.h>
#include <hip/hip_runtime.h>

typedef struct {
    float x, y, z; // Position
    float vx, vy, vz; // Velocity
} Particle;

__global__ void updateParticles(Particle *particles, int numParticles, float deltaTime) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numParticles) {
        // Simple physics update
        particles[idx].vy -= 9.81f * deltaTime; // Gravity effect

        // Update position
        particles[idx].x += particles[idx].vx * deltaTime;
        particles[idx].y += particles[idx].vy * deltaTime;
        particles[idx].z += particles[idx].vz * deltaTime;
    }
}

void saveParticlePositions(Particle *particles, int numParticles, FILE *file) {
    for (int i = 0; i < numParticles; i++) {
        fprintf(file, "%f %f %f\n", particles[i].x, particles[i].y, particles[i].z);
    }
    fprintf(file, "\n"); // Separate different frames by a newline
}

int main() {
    int numParticles = 1000;
    float deltaTime = 0.01f; // Time step for simulation
    int numIterations = 100; // Number of simulation steps

    Particle *particles_host = (Particle*)malloc(numParticles * sizeof(Particle));
    Particle *particles_device;

    // Initialize particles
    // ...

    hipMalloc((void**)&particles_device, numParticles * sizeof(Particle));
    hipMemcpy(particles_device, particles_host, numParticles * sizeof(Particle), hipMemcpyHostToDevice);

    FILE *file = fopen("particle_positions.txt", "w");
    if (file == NULL) {
        fprintf(stderr, "Failed to open the file for writing.\n");
        exit(1);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;

    for (int iter = 0; iter < numIterations; iter++) {
        updateParticles<<<blocksPerGrid, threadsPerBlock>>>(particles_device, numParticles, deltaTime);
        hipMemcpy(particles_host, particles_device, numParticles * sizeof(Particle), hipMemcpyDeviceToHost);
        saveParticlePositions(particles_host, numParticles, file);
    }

    fclose(file);
    hipFree(particles_device);
    free(particles_host);

    return 0;
}
